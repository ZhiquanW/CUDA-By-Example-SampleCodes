#include "hip/hip_runtime.h"
#include "../book.h"
#include <iostream>
__global__ void add(int a, int b, int *c);

int main(void) {
  int c;
  int *dev_c;

  HANDLE_ERROR(hipMalloc((void **)&dev_c, sizeof(int)));
  add<<<1, 1>>>(2, 7, dev_c);
  HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
  printf("2 + 7 = %d \n", c);
  hipFree(dev_c);
  return 0;
}
__global__ void add(int a, int b, int *c) { *c = a + b; }