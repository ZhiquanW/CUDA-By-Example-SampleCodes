#include "hip/hip_runtime.h"
#include "../book.h"
#include "pngmaster.h"
#include <cmath>
__global__ void gen_image(unsigned char *data);

#define IMAGE_DIM 640
int main() {
  const unsigned int image_height = IMAGE_DIM;
  const unsigned int image_width = IMAGE_DIM;
  pngmaster png_image(image_height, image_width);
  unsigned char *d_image_data;
  HANDLE_ERROR(hipMalloc((void **)&d_image_data, png_image.size_bytes()));

  dim3 block_size(16, 16);
  dim3 grid_size(IMAGE_DIM / block_size.x, IMAGE_DIM / block_size.y);
  gen_image<<<grid_size, block_size>>>(d_image_data);
  printf("%d \n", png_image.size_bytes());
  HANDLE_ERROR(hipMemcpy(png_image.data, d_image_data, png_image.size_bytes(),
                          hipMemcpyDeviceToHost));

  png_image.output("ripple.png");
  hipFree(d_image_data);
}

__global__ void gen_image(unsigned char *data) {
  int col = blockDim.x * blockIdx.x + threadIdx.x;
  int row = blockDim.y * blockIdx.y + threadIdx.y;
  int offset = row * gridDim.x * blockDim.x + col;
  printf("%d \n", offset);
  float fx = col - IMAGE_DIM / 2;
  float fy = row - IMAGE_DIM / 2;
  float dis = sqrtf(fx * fx + fy * fy);

  unsigned char grey =
      (unsigned char)(128.0f + 127.0f * cos(dis / 10.0f - 12 / 7.0f) /
                                   (dis / 10.0f + 1.0f));
  data[offset * 4 + 0] = 64;
  data[offset * 4 + 1] = grey;
  data[offset * 4 + 2] = 128;
  data[offset * 4 + 3] = 255;
}