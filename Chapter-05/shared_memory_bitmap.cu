#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_bitmap.h"
#include "pngmaster.h"
#define DIM 1024
#define PI 3.1415926535897932f
struct hipComplex {
    float r;
    float i;
    __device__ hipComplex(float a, float b) : r(a), i(b) {}
    __device__ float magnitude2(void) { return r * r + i * i; }
    __device__ hipComplex operator*(const hipComplex &a) { return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i); }
    __device__ hipComplex operator+(const hipComplex &a) { return hipComplex(r + a.r, i + a.i); }
};

__global__ void kernel(unsigned char *ptr) {
    // map from threadIdx/BlockIdx to pixel position
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = x + y * gridDim.x * blockDim.x;

    __shared__ float cache[16][16];
    // now calculate the value at that position
    const float period = 128.0f;
    cache[threadIdx.x][threadIdx.y] =
        255 * (sinf(x * 2.0f * PI / period) + 1.0f) * (sinf(y * 2.0f * PI / period) + 1.0f) / 4.0f;
    __syncthreads();
    ptr[offset * 4 + 0] = 0;
    ptr[offset * 4 + 1] = cache[15 - threadIdx.x][15 - threadIdx.y];
    ptr[offset * 4 + 2] = 0;
    ptr[offset * 4 + 3] = 255;
}
int main(void) {
    pngmaster bitmap(DIM, DIM);
    unsigned char *dev_bitmap;
    HANDLE_ERROR(hipMalloc((void **)&dev_bitmap, sizeof(unsigned char) * bitmap.height * bitmap.width * 4));
    dim3 grids(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    kernel<<<grids, threads>>>(dev_bitmap);
    HANDLE_ERROR(hipMemcpy(bitmap.data, dev_bitmap, sizeof(unsigned char) * bitmap.height * bitmap.width * 4,
                            hipMemcpyDeviceToHost));
    bitmap.output("1234.png");
    hipFree(dev_bitmap);
}
