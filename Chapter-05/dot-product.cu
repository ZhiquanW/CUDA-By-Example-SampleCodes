#include "hip/hip_runtime.h"
#include "../book.h"
#include <cmath>
#include <stdio.h>
const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blockPerGrid = min(32, (N + threadsPerBlock - 1) / threadsPerBlock);
__global__ void dot(float *, float *, float *);
int main() {
  float *a, *b, *partial_c;
  float *dev_a, *dev_b, *dev_partial_c;
  a = new float[N];
  b = new float[N];
  partial_c = new float[blockPerGrid];
  printf("%d %d\n", blockPerGrid, threadsPerBlock);
  HANDLE_ERROR(hipMalloc((void **)&dev_a, N * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&dev_b, N * sizeof(float)));
  HANDLE_ERROR(
      hipMalloc((void **)&dev_partial_c, blockPerGrid * sizeof(float)));
  for (int i = 0; i < N; ++i) {
    a[i] = i;
    b[i] = 0;
  }
  HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice));
  dot<<<blockPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);
  HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c,
                          blockPerGrid * sizeof(float),
                          hipMemcpyDeviceToHost));

  float final_result = 0;
  for (int i = 0; i < blockPerGrid; ++i) {
    final_result += partial_c[i];
  }
  printf("Final Result: %f \n", final_result);
}
__global__ void dot(float *a, float *b, float *c) {
  __shared__ float cache[threadsPerBlock];
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int cacheIndex = threadIdx.x;

  float tmp = 0;
  while (tid < N) {
    tmp += a[tid] * b[tid];

    tid += gridDim.x * blockDim.x;
  }
  cache[cacheIndex] = tmp;

  __syncthreads();
  int tmp_len = blockDim.x / 2;
  while (tmp_len != 0) {
    if (cacheIndex < tmp_len) {
      cache[cacheIndex] += cache[cacheIndex + tmp_len];
    }
    __syncthreads();
    tmp_len /= 2;
  }
  if (cacheIndex == 0) {
    c[blockIdx.x] = cache[0];
  }
}
