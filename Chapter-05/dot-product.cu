#include "hip/hip_runtime.h"
#include "../book.h"
#include <cmath>
#include <stdio.h>
const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = min(32, (N + threadsPerBlock - 1) / threadsPerBlock);
__global__ void dot(float *, float *, float *);
int main() {
  float *a, *b, *partial_c;
  float *dev_a, *dev_b, *dev_partial_c;
  a = new float[N];
  b = new float[N];
  partial_c = new float[blocksPerGrid];
  printf("%d %d\n", blocksPerGrid, threadsPerBlock);
  HANDLE_ERROR(hipMalloc((void **)&dev_a, N * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&dev_b, N * sizeof(float)));
  HANDLE_ERROR(
      hipMalloc((void **)&dev_partial_c, blocksPerGrid * sizeof(float)));
  for (int i = 0; i < N; ++i) {
    a[i] = i;
    b[i] = 0;
  }
  HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice));
  dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);
  HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c,
                          blocksPerGrid * sizeof(float),
                          hipMemcpyDeviceToHost));

  float final_result = 0;
  for (int i = 0; i < blocksPerGrid; ++i) {
    final_result += partial_c[i];
  }
  printf("Final Result: %.6f\n", final_result);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_partial_c);
  free(a);
  free(b);
  free(partial_c);
}
__global__ void dot(float *a, float *b, float *c) {
  __shared__ float cache[threadsPerBlock];
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int cacheIndex = threadIdx.x;

  float tmp = 0;
  while (tid < N) {
    tmp += a[tid] * b[tid];

    tid += gridDim.x * blockDim.x;
  }
  cache[cacheIndex] = tmp;

  __syncthreads();
  int tmp_len = blockDim.x / 2;
  while (tmp_len != 0) {
    if (cacheIndex < tmp_len) {
      cache[cacheIndex] += cache[cacheIndex + tmp_len];
    }
    __syncthreads();
    tmp_len /= 2;
  }
  if (cacheIndex == 0) {
    c[blockIdx.x] = cache[0];
  }
}
